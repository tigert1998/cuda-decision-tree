#include <vector>

#include "decision_tree.cuh"

void AssertCudaCorrect() {
  auto err = hipGetLastError();
  if (err != hipSuccess) {
    printf("%s\n", hipGetErrorString(err));
    exit(1);
  }
}

int main() {
  int8_t* samples_ptr;
  int8_t* targets_ptr;
  int8_t* dims_ptr;
  int8_t* vals_ptr;
  int8_t* bins_ptr;

  int num_codebooks, num_samples, vector_length, num_targets, dt_depth;

  std::cin >> num_codebooks >> num_samples >> vector_length >> num_targets >>
      dt_depth;

  std::vector<int8_t> samples(num_codebooks * num_samples * vector_length);
  std::vector<int8_t> targets(num_codebooks * num_samples);

  for (int i = 0; i < samples.size(); i++) {
    int x;
    std::cin >> x;
    samples[i] = x;
  }
  for (int i = 0; i < targets.size(); i++) {
    int x;
    std::cin >> x;
    targets[i] = x;
  }

  hipDeviceSetLimit(hipLimitMallocHeapSize, 1 << 30);

  hipMalloc(&samples_ptr, num_codebooks * num_samples * vector_length);
  hipMalloc(&targets_ptr, num_codebooks * num_samples);
  hipMalloc(&dims_ptr, num_codebooks * ((1 << dt_depth) - 1));
  hipMalloc(&vals_ptr, num_codebooks * ((1 << dt_depth) - 1));
  hipMalloc(&bins_ptr, num_codebooks * (1 << dt_depth));

  hipMemcpy(samples_ptr, samples.data(), samples.size(),
             hipMemcpyHostToDevice);
  hipMemcpy(targets_ptr, targets.data(), targets.size(),
             hipMemcpyHostToDevice);

  AssertCudaCorrect();

  ConstructDecisionTree<int8_t, int8_t, int8_t><<<num_codebooks, 1>>>(
      num_codebooks, num_samples, vector_length, num_targets, dt_depth,
      samples_ptr, targets_ptr, dims_ptr, vals_ptr, bins_ptr);

  AssertCudaCorrect();

  std::vector<int8_t> dims(num_codebooks * ((1 << dt_depth) - 1)),
      vals(num_codebooks * ((1 << dt_depth) - 1)),
      bins(num_codebooks * (1 << dt_depth));

  hipMemcpy(dims.data(), dims_ptr, dims.size(), hipMemcpyDeviceToHost);
  hipMemcpy(vals.data(), vals_ptr, vals.size(), hipMemcpyDeviceToHost);
  hipMemcpy(bins.data(), bins_ptr, bins.size(), hipMemcpyDeviceToHost);

  for (int i = 0; i < dims.size(); i++) printf("%d ", dims[i]);
  puts("");
  for (int i = 0; i < vals.size(); i++) printf("%d ", vals[i]);
  puts("");
  for (int i = 0; i < bins.size(); i++) printf("%d ", bins[i]);
  puts("");

  return 0;
}